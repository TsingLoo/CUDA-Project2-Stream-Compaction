#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define BLOCK_SIZE 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernWorkEfficientUpSweep(int paddedN, int n, int d, int* idata) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

			int halfStride = 1 << (d);
            int fullStride = halfStride * 2;

            if (k >= n || ((k + 1) % fullStride != 0)) {
                return;
            }

            // up-sweep
            idata[k] += idata[k - halfStride];
        }

        __global__ void setLastElementToZero(int paddedN, int* idata)
        {
            idata[paddedN - 1] = 0;
        }

        __global__ void kernWorkEfficientDownSweep(int paddedN, int n, int d, int* idata) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

            int halfStride = 1 << (d);
            int fullStride = halfStride * 2;

            if (k >= paddedN || ((k + 1) % fullStride != 0)) {
                return;
            }

            // down-sweep
			int originalLeftChildValue = idata[k - halfStride];
			int parentValue = idata[k];

			idata[k - halfStride] = parentValue ;
			idata[k] = parentValue + originalLeftChildValue;
        }

        void scanWithoutTimer(int n, int* odata, const int* idata) {
            int paddedN = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid((paddedN + BLOCK_SIZE - 1) / BLOCK_SIZE);

            int* dev_idata;

            hipMalloc((void**)&dev_idata, paddedN * sizeof(int));
            hipMemset(dev_idata, 0, paddedN * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            for (int d = 0; d <= ilog2ceil(n) - 1; d++) {
                kernWorkEfficientUpSweep << <fullBlocksPerGrid, BLOCK_SIZE >> > (paddedN, n, d, dev_idata);
            }

            setLastElementToZero << <1, 1 >> > (paddedN, dev_idata);

            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                kernWorkEfficientDownSweep << <fullBlocksPerGrid, BLOCK_SIZE >> > (paddedN, n, d, dev_idata);
            }

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            scanWithoutTimer(n, odata, idata);
            timer().endGpuTimer();
        }

        __global__ void kernComputeCompactFlag(int n, int *odata, const int* idata) 
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
            if (idx >= n) {
                return;
            }

            if (idata[idx] != 0) {
                odata[idx] = 1;
            }
            else {
                odata[idx] = 0;
			}
        }

        __global__ void kernCompactScatter(int n, int* odata, const int* dev_idata, const int* flags, const int* scanResult)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            if (idx >= n || flags[idx] == 0) {
                return;
            }

			odata[scanResult[idx]] = dev_idata[idx];
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            int* dev_idata;
            int* dev_flags;
            int* dev_scanResult;
            int* dev_odata;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_flags, n * sizeof(int));
            hipMalloc((void**)&dev_scanResult, n * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            //compute the temporary array
			kernComputeCompactFlag<< <fullBlocksPerGrid, BLOCK_SIZE >> > (n, dev_flags, dev_idata);
            hipMemcpy(odata, dev_flags, n * sizeof(int), hipMemcpyDeviceToHost);

			//compute the scan of the temporary array
            int* host_scanResult = new int[n];
            scanWithoutTimer(n, host_scanResult, odata);
            hipMemcpy(dev_scanResult, host_scanResult, n * sizeof(int), hipMemcpyHostToDevice);

			//compute the scatter
            int totalCount = host_scanResult[n - 1] + odata[n - 1];
            hipMalloc((void**)&dev_odata, totalCount * sizeof(int));
            kernCompactScatter<< <fullBlocksPerGrid, BLOCK_SIZE >> > (n, dev_odata, dev_idata, dev_flags, dev_scanResult);
            
            hipMemcpy(odata, dev_odata, totalCount * sizeof(int), hipMemcpyDeviceToHost);


            hipFree(dev_flags);

            timer().endGpuTimer();
            return totalCount;
        }
    }
}
